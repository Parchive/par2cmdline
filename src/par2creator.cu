#include "hip/hip_runtime.h"
#include "libpar2internal.h"
#include <iostream>

#ifdef _MSC_VER
#ifdef _DEBUG
#undef THIS_FILE
static char THIS_FILE[]=__FILE__;
#define new DEBUG_NEW
#endif
#endif

// ProcessData, but on CUDA device.
bool Par2Creator::ProcessDataCu()
{
  // Start at an offset of 0 within a block.
  // Continue until the end of the block.
  u64 blockOffset = 0;
  while (blockOffset < blocksize) {
    // Work out how much data to process this time.
    size_t blockLen = (size_t) min((u64) chunksize, blocksize - blockOffset);

    // Clear the output buffer
    memset(outputbuffer, 0, chunksize * recoveryblockcount);

    // If we have deferred computation of the file hash and block crc and hashes
    // sourcefile and sourceindex will be used to update them during
    // the main recovery block computation
    vector<Par2CreatorSourceFile*>::iterator sourcefile = sourcefiles.begin();
    u32 sourceindex = 0;

    vector<DataBlock>::iterator sourceblock;
    u32 inputIdx;

    DiskFile *lastopenfile = NULL;

    // Read blockLen bytes of each input block into inputbuffer
    for ((sourceblock=sourceblocks.begin()),(inputIdx=0);
        sourceblock != sourceblocks.end();
        ++sourceblock, ++inputIdx)
    {
      // Are we reading from a new file?
      if (lastopenfile != (*sourceblock).GetDiskFile())
      {
        // Close the last file
        if (lastopenfile != NULL)
        {
          lastopenfile->Close();
        }

        // Open the new file
        lastopenfile = (*sourceblock).GetDiskFile();
        if (!lastopenfile->Open())
        {
          return false;
        }
      }

      // Read data from the current input block
      if (!sourceblock->ReadData(blockOffset, blockLen, &((u8*) inputbuffer)[blockLen * inputIdx]))
        return false;

      if (deferhashcomputation)
      {
        assert(blockOffset == 0 && blockLen == blocksize);
        assert(sourcefile != sourcefiles.end());

        (*sourcefile)->UpdateHashes(sourceindex, &((u8*) inputbuffer)[blockLen * inputIdx], blockLen);
      }

      // Work out which source file the next block belongs to
      if (++sourceindex >= (*sourcefile)->BlockCount())
      {
        sourceindex = 0;
        ++sourcefile;
      }
    }

    // Close the last file
    if (lastopenfile != NULL)
    {
      lastopenfile->Close();
    }

    // Process the data through the RS matrix on GPU
    std::cout << "blockLen: " << blockLen << "; Sourceblockcount: " << sourceblockcount << " Recoveryblockcount: " << recoveryblockcount << std::endl;
    if (!rs.ProcessCu(blockLen, 0, sourceblockcount - 1, inputbuffer, 0, recoveryblockcount - 1, outputbuffer)) {
      return false;
    }

    if (noiselevel > nlQuiet)
    {
      // Update a progress indicator
      u32 oldfraction = (u32)(1000 * progress / totaldata);
      progress += blockLen * sourceblockcount * recoveryblockcount;
      u32 newfraction = (u32)(1000 * progress / totaldata);

      if (oldfraction != newfraction)
      {
        sout << "Processing: " << newfraction/10 << '.' << newfraction%10 << "%\r" << flush;
      }
    }

    // For each output block
    for (u32 outputblock=0; outputblock<recoveryblockcount;outputblock++)
    {
      // Select the appropriate part of the output buffer
      u8 *outbuf = &((u8*) outputbuffer)[chunksize * outputblock];

      // Write the data to the recovery packet
      if (!recoverypackets[outputblock].WriteData(blockOffset, blockLen, outbuf))
        return false;
    }

    if (noiselevel > nlQuiet)
      sout << "Wrote " << recoveryblockcount * blockLen << " bytes to disk" << endl;

    blockOffset += blockLen;
  }

  return true;
}